#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : katsevich_backprojection.cu
 Author      : Rui Liu
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <vector>
#include <algorithm>
//#include "KatsevichBackprojection.hpp"
#define TWOPI (6.283185307179586)
#define INV_TWOPI (0.1590250231624044)
#define PI (3.141592653589793)

/**
* This macro checks return value of the CUDA runtime call and exits
* the application if the call failed.
*/
#if DEBUG
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
// Same function as CUDA_CHECK_RETURN
#define CUDA_SAFE_CALL(call) do{ hipError_t err = call; if (hipSuccess != err) {  fprintf (stderr, "Cuda error in file '%s' in line %i : %s.", __FILE__, __LINE__, hipGetErrorString(err) );  exit(EXIT_FAILURE);  } } while (0)
#else
#define CUDA_CHECK_RETURN(value) {value;}
#define CUDA_SAFE_CALL(value) {value;}
#endif

#ifndef nullptr
#define nullptr NULL
#endif

#ifndef EPSILON
#define EPSILON (0.0000001)
#endif




//Create texture object and corresponding hipArray function
template<typename T>
void createTextureObject(
	hipTextureObject_t& texObj, //return: texture object pointing to the hipArray
	hipArray* d_prjArray, // return: hipArray storing the data
	int Width, int Height, int Depth, // data size
	T* sourceData, // where is the data
	hipMemcpyKind memcpyKind, // data from host or memory
	hipTextureAddressMode addressMode, // how to address the texture (clamp, border ...)
	hipTextureFilterMode textureFilterMode, // usually linear filtering (double --> int2 use pointer not linear interpolation)
	hipTextureReadMode textureReadMode, // usually use element wise reading mode.
	bool isNormalized) // usually false
{
	hipExtent prjSize;
	prjSize.width = Width;
	prjSize.height = Height;
	prjSize.depth = Depth;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>();
	hipMalloc3DArray(&d_prjArray, &channelDesc, prjSize);
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(
		(void*)sourceData, prjSize.width * sizeof(T),
		prjSize.width, prjSize.height);
	copyParams.dstArray = d_prjArray;
	copyParams.extent = prjSize;
	copyParams.kind = memcpyKind;
	hipMemcpy3D(&copyParams);
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = d_prjArray;
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = addressMode;
	texDesc.addressMode[1] = addressMode;
	texDesc.addressMode[2] = addressMode;
	texDesc.filterMode = textureFilterMode;
	texDesc.readMode = textureReadMode;

	texDesc.normalizedCoords = isNormalized;
	CUDA_SAFE_CALL(hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr));
}


// Destroy a GPU array and corresponding TextureObject
void destroyTextureObject(hipTextureObject_t& texObj, hipArray* d_array)
{
	hipDestroyTextureObject(texObj);
	hipFreeArray(d_array);
}




template<typename T>
__device__ __host__ inline void PISegment(const T x, const T y, const T z, T& BAngle, T& TAngle)
{
	T delta = 1.0;
	T bmax = z * TWOPI;
	T bmin = bmax - TWOPI;

	T r2 = x * x + y * y;

	T sb = 0;
	T st = 0;

	while ((bmax - bmin > EPSILON) && (delta > EPSILON))
	{
		sb = (bmax + bmin) * 0.5;
		T sinsb = sin(sb);
		T cossb = cos(sb);
		T tempcos = 2.0 * (1.0 - y * sinsb - x * cossb);
		assert(tempcos != 0);
		T t = (1.0 - r2) / tempcos;
		T templan = acos((y * cossb - x * sinsb) / sqrt(tempcos + r2 - 1.0));
		st = 2.0 * templan + sb;
		T zz = (sb * t + (1.0 - t) * st) * INV_TWOPI;
		if (zz < z)
		{
			bmin = sb;
		}
		else
		{
			bmax = sb;
		}
		delta = fabs(zz - z);
	}

	BAngle = sb;
	TAngle = st;
}


// Note: this projection do not consider the edging situation for backprojection
__global__ void backProjectionKer(
	float* Image,
	hipTextureObject_t ProjTex,
	int RecMX, int RecMY, int RecMZ,
	float ObjRSquare,
	float* __restrict__ xCor, float* __restrict__ yCor, float* __restrict__ zCor,
	float ScanR,
	float StdDis,
	float DeltaU, float DeltaV,
	float HalfY, float HalfZ,
	float HelicP,
	float DeltaL,
	float ProjCtr,
	int ProjBeginIndex,
	int ProjNum,
	int ProjScale)
{
	int Zindex = threadIdx.x + blockIdx.x * blockDim.x;
	int Xindex = threadIdx.y + blockIdx.y * blockDim.y;
	int Yindex = threadIdx.z + blockIdx.z * blockDim.z;
	const size_t idx = (Xindex * RecMY + Yindex) * RecMZ + Zindex;
	if (Xindex < RecMX && Yindex < RecMY && Zindex < RecMZ)
	{
		const float X = xCor[Xindex];
		const float Y = yCor[Yindex];
		const float Z = zCor[Zindex];
		if (pow(X, 2.0f) + pow(Y, 2.0f) >= ObjRSquare)
			return;

		double BAngle;
		double TAngle;
		PISegment<double>(X / ScanR, Y / ScanR, Z / HelicP, BAngle, TAngle);
		BAngle = BAngle / DeltaL + ProjCtr - ProjBeginIndex;
		TAngle = TAngle / DeltaL + ProjCtr - ProjBeginIndex;

		int Bindex = int(BAngle);
		int Tindex = ceil(TAngle);
		if (Bindex < 0)
		{
			Bindex = 0;
		}
		if (Bindex > ProjNum - 1)
		{
			Bindex = ProjNum - 1;
		}
		if (Tindex < 0)
		{
			Tindex = 0;
		}
		if (Tindex > ProjNum - 1)
		{
			Tindex = ProjNum - 1;
		}
		float tpdata = 0.0f;
		for (int ProjIndex = Bindex; ProjIndex <= Tindex; ProjIndex++)
		{
			float theta = (ProjIndex + ProjBeginIndex - ProjCtr) * DeltaL;
			float cost = cosf(theta);
			float sint = sinf(theta);

			float DPSx = X - ScanR * cost;
			float DPSy = Y - ScanR * sint;
			float DPSz = Z - HelicP * theta * INV_TWOPI;
			float factor = sqrtf(DPSx * DPSx + DPSy * DPSy + DPSz * DPSz);
			float fenmu = -(DPSx * cost + DPSy * sint);
			float YY = DPSy * cost - DPSx * sint;
			YY = YY * StdDis / (fenmu * DeltaU) + HalfY;
			float ZZ = DPSz * StdDis / (fenmu * DeltaV) + HalfZ;
			float temp = tex3D<float>(ProjTex, YY + 0.5f, ZZ + 0.5f, ProjIndex + 0.5f);
			tpdata += temp / factor;
		}
		tpdata = -tpdata / ProjScale;
		Image[idx] = tpdata;

	}
}



void backProjection(
	thrust::host_vector<float>& hImage,
	thrust::host_vector<float>& hProj,
	int RecMX, int RecMY, int RecMZ,
	float ObjRSquare,
	const thrust::host_vector<float>& hxCor,
	const thrust::host_vector<float>& hyCor,
	const thrust::host_vector<float>& hzCor,
	float ScanR,
	float StdDis,
	float DeltaU, float DeltaV,
	float HalfY, float HalfZ,
	int YL, int YLZL,
	float HelicP,
	float DeltaL,
	float ProjCtr,
	float ProjBeginIndex,
	int ProjNum, // number of projections
	int ProjScale,
	int threadidx, int threadidy, int threadidz)
{
	thrust::device_vector<float> Image = hImage;
	thrust::device_vector<float> xCor = hxCor;
	thrust::device_vector<float> yCor = hyCor;
	thrust::device_vector<float> zCor = hzCor;

	dim3 blk(threadidx, threadidy, threadidz);
	dim3 gid(
		(RecMZ + blk.x - 1) / blk.x,
		(RecMX + blk.y - 1) / blk.y,
		(RecMY + blk.z - 1) / blk.z);
	int ZL = YLZL / YL;
	hipTextureObject_t projTex;
	hipArray* d_projArray = nullptr;
	createTextureObject<float>(projTex, d_projArray,
		YL, ZL, ProjNum,
		&(hProj[0]),
		hipMemcpyHostToDevice,
		hipAddressModeClamp,
		hipFilterModeLinear,
		hipReadModeElementType, false);

	backProjectionKer<< <gid, blk >> > (
		thrust::raw_pointer_cast(&Image[0]),
		projTex,
		RecMX, RecMY, RecMZ, ObjRSquare,
		thrust::raw_pointer_cast(&xCor[0]),
		thrust::raw_pointer_cast(&yCor[0]),
		thrust::raw_pointer_cast(&zCor[0]),
		ScanR, StdDis, DeltaU, DeltaV, HalfY, HalfZ,
		HelicP, DeltaL, ProjCtr, ProjBeginIndex,
		ProjNum, ProjScale);
	hImage = Image;
	destroyTextureObject(projTex, d_projArray);
}

extern "C"
void backProjection(
	double* hImage, double* hProj,
	int RecMX, int RecMY, int RecMZ,
	double ObjRSquare,
	double* hxCor, double* hyCor, double* hzCor,
	double ScanR,
	double StdDis,
	double DeltaU, double DeltaV,
	double HalfY, double HalfZ,
	int YL, int YLZL,
	double HelicP, double DeltaL, double ProjCtr,
	int ProjBeginIndex,
	int ProjNum, // number of projections
	int ProjScale,
	int threadidx, int threadidy, int threadidz)
{
	thrust::host_vector<float> Image(RecMX * RecMY * RecMZ, 0.0f);
	thrust::host_vector<float> Proj(ProjNum * YLZL, 0.0f);
	thrust::host_vector<float> xCor(ProjNum, 0.0f);
	thrust::host_vector<float> yCor(ProjNum, 0.0f);
	thrust::host_vector<float> zCor(ProjNum, 0.0f);

	thrust::fill(Image.begin(), Image.end(), 0.0f);

	thrust::copy(hProj, hProj + ProjNum * YLZL, &(Proj[0]));
	thrust::copy(hxCor, hxCor + RecMX, &(xCor[0]));
	thrust::copy(hyCor, hyCor + RecMY, &(yCor[0]));
	thrust::copy(hzCor, hzCor + RecMZ, &(zCor[0]));

	backProjection(Image, Proj, RecMX, RecMY, RecMZ, ObjRSquare,
		xCor, yCor, zCor,
		ScanR, StdDis, DeltaU, DeltaV, HalfY, HalfZ, YL, YLZL,
		HelicP, DeltaL, ProjCtr, ProjBeginIndex, ProjNum,
		ProjScale, threadidx, threadidy, threadidz);
	thrust::copy(&(Image[0]), &(Image[0]) + RecMX * RecMY * RecMZ, hImage);
}
